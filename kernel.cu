
#include <hip/hip_runtime.h>
#define TILE_SIZE 32

__global__ void calc_account(int *changes, int *account, int *sum, int clients, int periods) {
    int ty = threadIdx.y;
    int tx = threadIdx.x;
	
    int row = blockIdx.y * TILE_SIZE + ty;
    int col = blockIdx.x * TILE_SIZE + tx;

    __shared__ int tile[TILE_SIZE][TILE_SIZE];

    // load data into shared memory
    if (row < periods && col < clients) {
        tile[ty][tx] = changes[row * clients + col];
    } else {
        tile[ty][tx] = 0;
    }
    __syncthreads();

    account[row * clients + col] = tile[ty][tx];
}

__global__ void calc_sum(int *account, int *sum, int clients, int periods) {
    // partial sum within one block (one row)
    __shared__ int partial_sum[128];

    int period = blockIdx.x;
    int tid = threadIdx.x;

    partial_sum[tid] = 0;

    for (int col = tid; col < clients; col += blockDim.x) {
        partial_sum[tid] += account[period * clients + col];  // Accumulate sum for this thread's chunk
    }
    __syncthreads();

    // Perform parallel reduction to sum the row elements using shared memory
    for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        if (tid < stride) {
            partial_sum[tid] += partial_sum[tid + stride];
        }
        __syncthreads(); // Ensure all threads have completed their work before proceeding
    }

    // The first thread in the block writes the result to the output row sum
    if (tid == 0) {
        sum[period] = partial_sum[0];  // Final sum for this row
    }
}

void solveGPU(int *changes, int *account, int *sum, int clients, int periods) {
    int BLOCK_SIZE = TILE_SIZE * TILE_SIZE; 
    int N_BLOCKS = (clients / TILE_SIZE) * (periods / TILE_SIZE);

    calc_account<<<N_BLOCKS, BLOCK_SIZE>>>(changes, account, sum, clients, periods);

    BLOCK_SIZE = 128;
    N_BLOCKS = periods;

    calc_sum<<<N_BLOCKS, BLOCK_SIZE>>>(account, sum, clients, periods);
}
