
#include <hip/hip_runtime.h>
#define TILE_SIZE_X 32
#define TILE_SIZE_Y 32

__global__ void calc_account(int *changes, int *account, int *sum, int clients, int periods, int tile_y) {
    int ty = threadIdx.y;
    int tx = threadIdx.x;
	
    int row = tile_y * TILE_SIZE_Y + ty;
    int col = blockIdx.x * TILE_SIZE_X + tx;

    __shared__ int tile[TILE_SIZE_Y][TILE_SIZE_X];
    int prev_block_val = tile_y == 0 ? 0 : account[(row - 1) * clients + col];

    // load data into shared memory
    if (row < periods && col < clients) {
        tile[ty][tx] = changes[row * clients + col] + prev_block_val;
    } else { // remove unnecessary else?
        tile[ty][tx] = 0;
    }
    __syncthreads();

    for (int stride = 1; stride < TILE_SIZE_Y; stride *= 2) {
        int val = ty >= stride ? tile[ty - stride][tx] : 0;
        __syncthreads();
        tile[ty][tx] += val;
        __syncthreads();
    }

    account[row * clients + col] = tile[ty][tx];
}

__global__ void calc_sum__parallel(int *account, int *sum, int clients, int periods) {
    // partial sum within one block (one row)
    __shared__ int partial_sum[128];

    int period = blockIdx.x;
    int tid = threadIdx.x;

    partial_sum[tid] = 0;

    for (int col = tid; col < clients; col += blockDim.x) {
        partial_sum[tid] += account[period * clients + col];  // Accumulate sum for this thread's chunk
    }
    __syncthreads();

    // Perform parallel reduction to sum the row elements using shared memory
    for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        if (tid < stride) {
            partial_sum[tid] += partial_sum[tid + stride];
        }
        __syncthreads(); // Ensure all threads have completed their work before proceeding
    }

    // The first thread in the block writes the result to the output row sum
    if (tid == 0) {
        sum[period] = partial_sum[0];  // Final sum for this row
    }
}

__inline__ __device__ int warpReduceSum(int val) {
    // Reduce within a warp using shfl_down_sync
    for (int offset = warpSize >> 1; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__global__ void calc_sum__parallel_warp(int *account, int *sum, int clients, int periods) {
    int period = blockIdx.x;  // Each block handles one row (one period)
    int tid = threadIdx.x;    // Thread ID within the block

    int lane = tid % warpSize; // Lane within the warp
    int warpId = tid / warpSize; // Warp ID within the block

    // Shared memory for the partial sums from each warp
    __shared__ int warp_sums[32];  // Max 32 warps per block

    // Initialize the thread's local sum
    int local_sum = 0;

    // Each thread accumulates a portion of the row's elements
    for (int col = tid; col < clients; col += blockDim.x) {
        local_sum += account[period * clients + col];
    }

    // Perform warp-level reduction
    local_sum = warpReduceSum(local_sum);

    // Store the result of each warp's reduction in shared memory
    if (lane == 0) {
        warp_sums[warpId] = local_sum;
    }
    __syncthreads();

    // The first warp in the block reduces the partial sums from each warp
    if (warpId == 0) {
        local_sum = (tid < blockDim.x / warpSize) ? warp_sums[lane] : 0;
        if (lane == 0) {
            for (int i = 1; i < blockDim.x / warpSize; i++) {
                local_sum += warp_sums[i];
            }
            sum[period] = local_sum; // Write the final sum to the output
        }
    }
}

void solveGPU(int *changes, int *account, int *sum, int clients, int periods) {
    dim3 blockDim(TILE_SIZE_X, TILE_SIZE_Y);
    dim3 gridDim(clients / blockDim.x);

    for (int i = 0; i < periods / TILE_SIZE_Y; i++)
        calc_account<<<gridDim, blockDim>>>(changes, account, sum, clients, periods, i);

    int BLOCK_SIZE = 128;
    int N_BLOCKS = periods;

    calc_sum__parallel<<<N_BLOCKS, BLOCK_SIZE>>>(account, sum, clients, periods);
}
