#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// nvcc -o framework framework.cu

#include "kernel.cu"
#include "kernel_CPU.C"

void print_matrices(int *cpu, int *gpu, int rows, int cols) {
    printf("\n");
    printf("Correct matrix:\n\n");

    for (int i = 0; i < rows; i++) {
	for (int j = 0; j < cols; j++) {
	    printf("%d ", cpu[i * cols + j]);
	}

	printf("|\n");
    }

    printf("\n");
    printf("GPU matrix:\n\n");

    for (int i = 0; i < rows; i++) {
	for (int j = 0; j < cols; j++) {
	    printf("%d ", gpu[i * cols + j]);
	}

	printf("|\n");
    }
}

void test_performance() {
    const int CLIENTS = 512;
    const int PERIODS = 512;

    // CPU data
    int *changes, *account, *sum, *account_gpu, *sum_gpu;
    changes = account = sum = account_gpu = sum_gpu = NULL;
    // GPU counterparts
    int *dchanges, *daccount, *dsum;
    dchanges = daccount = dsum = NULL;

    // create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate and set host memory
    changes = (int*)malloc(CLIENTS*PERIODS*sizeof(changes[0]));
    account = (int*)malloc(CLIENTS*PERIODS*sizeof(account[0]));
    sum = (int*)malloc(PERIODS*sizeof(sum[0]));
    account_gpu = (int*)malloc(CLIENTS*PERIODS*sizeof(account_gpu[0]));
    sum_gpu = (int*)malloc(PERIODS*sizeof(sum[0]));

    for (int i = 0; i < CLIENTS*PERIODS; i++)
        changes[i] = int(100.0f*(float)rand() / float(RAND_MAX));
 
    // allocate and set device memory
    if (hipMalloc((void**)&dchanges, CLIENTS*PERIODS*sizeof(dchanges[0])) != hipSuccess
    || hipMalloc((void**)&daccount, CLIENTS*PERIODS*sizeof(daccount[0])) != hipSuccess 
    || hipMalloc((void**)&dsum, PERIODS*sizeof(dsum[0])) != hipSuccess){
        fprintf(stderr, "Device memory allocation error!\n");
        goto cleanup;
    }
    hipMemcpy(dchanges, changes, CLIENTS*PERIODS*sizeof(dchanges[0]), hipMemcpyHostToDevice);
    hipMemset(daccount, 0, CLIENTS*PERIODS*sizeof(daccount[0]));
    hipMemset(dsum, 0, PERIODS*sizeof(dsum[0]));

    // solve on CPU
    printf("Solving on CPU...\n");
    hipEventRecord(start, 0);
    solveCPU(changes, account, sum, CLIENTS, PERIODS);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("CPU performance: %f megavalues/s\n",
        float(CLIENTS)*float(PERIODS)/time/1e3f);

    // solve on GPU
    printf("Solving on GPU...\n");
    hipEventRecord(start, 0);
    for(int i = 0; i < 100; i++) 
        solveGPU(dchanges, daccount, dsum, CLIENTS, PERIODS);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GPU performance: %f megavalues/s\n",
        float(CLIENTS)*float(PERIODS)/time/1e3f*100);

    // check GPU results
    hipMemcpy(account_gpu, daccount, CLIENTS*PERIODS*sizeof(daccount[0]), hipMemcpyDeviceToHost);
    for (int j = 0; j < PERIODS; j++)
        for (int i = 0; i < CLIENTS; i++)
            if (account[j*CLIENTS+i] != account_gpu[j*CLIENTS+i]) { 
                fprintf(stderr, "Account data mismatch at index %i, %i: %i should be %i :-(\n", i, j, account_gpu[j*CLIENTS+i], account[j*CLIENTS+i]);
                goto cleanup;
            }
    hipMemcpy(sum_gpu, dsum, PERIODS*sizeof(dsum[0]), hipMemcpyDeviceToHost);
    for (int i = 0; i < PERIODS; i++)
        if (sum[i] != sum_gpu[i]) {
            fprintf(stderr, "Sum data mismatch at index %i: %i should be %i :-(\n", i, sum_gpu[i], sum[i]);
                goto cleanup;
        }

    cleanup:
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // print_matrices(account, account_gpu, PERIODS, CLIENTS);
    // print_matrices(sum, sum_gpu, 1, PERIODS);

    if (dchanges) hipFree(dchanges);
    if (daccount) hipFree(daccount);
    if (dsum) hipFree(dsum);
    if (changes) free(changes);
    if (account) free(account);
    if (sum) free(sum);
    if (account_gpu) free(account_gpu);
    if (sum_gpu) free (sum_gpu);
}

int main(int argc, char **argv){
    // parse command line
    int device = 0;
    if (argc == 2) 
        device = atoi(argv[1]);
    if (hipSetDevice(device) != hipSuccess){
        fprintf(stderr, "Cannot set CUDA device!\n");
        exit(1);
    }
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Using device %d: \"%s\"\n", device, deviceProp.name);

    test_performance();
    

    return 0;
}
