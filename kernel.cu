
#include <hip/hip_runtime.h>
#define TILE_SIZE_X 32
#define TILE_SIZE_Y 32

__global__ void calc_account(int *changes, int *account, int *sum, int clients, int periods, int tile_y) {
    int ty = threadIdx.y;
    int tx = threadIdx.x;
	
    int row = tile_y * TILE_SIZE_Y + ty;
    int col = blockIdx.x * TILE_SIZE_X + tx;

    __shared__ int tile[TILE_SIZE_Y][TILE_SIZE_X];
    int prev_block_val = tile_y == 0 ? 0 : account[(row - 1) * clients + col];

    // Load data into shared memory
    if (row < periods && col < clients) {
        tile[ty][tx] = changes[row * clients + col] + prev_block_val;
    } else {
        tile[ty][tx] = 0;
    }
    __syncthreads();

    for (int stride = 1; stride < TILE_SIZE_Y; stride *= 2) {
        int val = ty >= stride ? tile[ty - stride][tx] : 0;
        __syncthreads();
        tile[ty][tx] += val;
        __syncthreads();
    }

    account[row * clients + col] = tile[ty][tx];
}

__global__ void calc_sum(int *account, int *sum, int clients, int periods) {
    // partial sum within one block (one row)
    __shared__ int partial_sum[128];

    int period = blockIdx.x;
    int tid = threadIdx.x;

    partial_sum[tid] = 0;

    for (int col = tid; col < clients; col += blockDim.x) {
        partial_sum[tid] += account[period * clients + col];  // Accumulate sum for this thread's chunk
    }
    __syncthreads();

    // Perform parallel reduction to sum the row elements using shared memory
    for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        if (tid < stride) {
            partial_sum[tid] += partial_sum[tid + stride];
        }
        __syncthreads(); // Ensure all threads have completed their work before proceeding
    }

    // The first thread in the block writes the result to the output row sum
    if (tid == 0) {
        sum[period] = partial_sum[0];  // Final sum for this row
    }
}

__global__ void calc_account_8192(int *changes, int *account, int *sum, int clients, int periods) {
    // __shared__ int tile[TILE_SIZE_Y][TILE_SIZE_X];
    int clientIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // int clientIdx = blockIdx.x;

    // shared_sum[localIdx] = 0;
    // __syncthreads();

    for (int j = 0; j < periods; j++) {
	int accountIdx = j * clients + clientIdx;
        int deposit = 0;

	if (j == 0) {
	    deposit = changes[accountIdx];
	} else {
	    deposit = account[(j - 1) * clients + clientIdx] + changes[accountIdx];
	}

	account[accountIdx] = deposit;
	atomicAdd(&sum[j], deposit);

	// atomicAdd(&shared_sum[j], account[accountIdx]);
    }
    __syncthreads();

    // if (threadIdx.x == 0)
	// atomicAdd(&sum[localIdx], shared_sum[localIdx]);
}

void solveGPU(int *changes, int *account, int *sum, int clients, int periods) {
    // if (clients == 8192 && periods == 8192) {
        dim3 blockDim(128);
	dim3 gridDim((clients + blockDim.x - 1) / blockDim.x);
	// dim3 gridDim(8192);

	calc_account_8192<<<gridDim, blockDim>>>(changes, account, sum, clients, periods);
    /*} else {
        dim3 blockDim(TILE_SIZE_X, TILE_SIZE_Y);
        dim3 gridDim(clients / blockDim.x);

        for (int i = 0; i < periods / TILE_SIZE_Y; i++)
            calc_account<<<gridDim, blockDim>>>(changes, account, sum, clients, periods, i);

        int BLOCK_SIZE = 128;
        int N_BLOCKS = periods;

        calc_sum<<<N_BLOCKS, BLOCK_SIZE>>>(account, sum, clients, periods);
    }*/
}
